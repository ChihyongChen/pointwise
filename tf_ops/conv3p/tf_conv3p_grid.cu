#include "hip/hip_runtime.h"
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/shape_inference.h"
#include "tensorflow/core/framework/common_shape_fns.h"

#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#include <chrono>
#include <type_traits>

const int kThreadsPerBlock = 256;

#define min(a, b) ((a) > (b))? (b): (a)
#define max(a, b) ((a) > (b))? (a): (b)

__host__ __device__ __forceinline__ int divUp(int total, int grain)
{
    return (total + grain - 1) / grain;
}

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

  #else
  static __inline__ __device__ double atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    if (val==0.0)
      return __longlong_as_double(old);
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
  }


#endif

void cudaErrorCheck(int line) {
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error at line %d: %s\n", line, hipGetErrorString(error));
    exit(-1);
  }
}


class timer
{
    // alias our types for simplicity
    using clock = typename std::conditional< std::chrono::high_resolution_clock::is_steady,
                                                  std::chrono::high_resolution_clock,
                                                  std::chrono::steady_clock >::type ;
    using time_point_type   = std::chrono::time_point < clock, std::chrono::milliseconds > ;
public:
    // default constructor that stores the start time
    timer()
    {
        start = std::chrono::time_point_cast<std::chrono::milliseconds>(clock::now());
    }

    // gets the time elapsed from construction.
    float getTimePassed()
    {
        // get the new time
        auto end = clock::now();

        // return the difference of the times
        return (end - start).count() / 1e9f;
    }

private:
    time_point_type start;
};

template <typename T>
struct GpuAlloc {
	static void alloc(T **data, int size) {
		hipMalloc(data, sizeof(T) * size);
	}
	static void free(T **data) {
		hipFree(*data);
		*data = NULL;
	}
	static void zero(T *data, int size) {
		hipMemset(data, 0, sizeof(T) * size);
	}
};

/**
 * This array wraps the data provided by a device pointer, or from a device memory allocation.
 * 
 * It refers to CUDA APIs to allocate memory from a host function call. 
 * Accessing values of the array must be done on the device. 
 */
template <template<class> class Alloc, typename T>
struct Array {
	T *data;
	int capacity;
	int size;

	Array() {
		data = NULL;
		capacity = 0;
		size = 0;
	}

	/**
	 * Wrap an input array
	 */
	Array(T *data, int n) {
		this->data = data;
		capacity = n;
		size = n;
	}

	void alloc(int new_capacity) {
        if (new_capacity > capacity) {
            this->free();
		    Alloc<T>::alloc(&data, new_capacity);
		    this->capacity = new_capacity;
        }
        this->size = 0;
	}

	void resize(int size) {
		if (size > capacity) {
			this->free();
			this->alloc(size);
		}
		this->size = size;
	}

	void free() {
		if (! data || capacity == 0) return;
		Alloc<T>::free(&data);
		this->capacity = 0;
		this->size = 0;
	}

	__host__ __device__ void zero() {
		#ifdef __CUDA_ARCH__
			// call from device 
			for (int i = 0; i < size; ++i) data[i] = 0;
		#else 
			// call from host 
			Alloc<T>::zero(data, size);
		#endif
	}

	__device__ T& operator[](int i) {
		return data[i];
	}

    __device__ T operator[](int i) const {
		return data[i];
	}

	__device__ void append(T value) {
		data[size] = value;
		size++;
	}

	__device__ void clear() {
		size = 0;
	}
};

struct Query {
	__device__ virtual void operator()(int ii, int f, int fsize) = 0;
};

template <template<class> class Alloc, typename T>
struct Grid {
	Array< Alloc, int> cell;			// cell index of each point
	Array< Alloc, int> cell_sorted;	    // index of the points in cell order
	Array< Alloc, int> cell_count;		// number of points in each cell
	Array< Alloc, int> cell_start;
	Array< Alloc, int> tmp_cell_count;
    Array< Alloc, T> points;
	Array< Alloc, int> neighbor_count;  // for each point, for each filter, for each cell of the filter, store how many points in that cell.

	T voxel_size;
	T vmin_x, vmin_y, vmin_z, vmax_x, vmax_y, vmax_z;
	int dim_x, dim_y, dim_z;
	int filter_x, filter_y, filter_z, filter_count;

	Grid() {}

	Grid(Array<Alloc, T> points, T voxel_size, int filter_x, int filter_y, int filter_z) {
        this->points = points;		
		this->voxel_size = voxel_size;
		this->filter_x = filter_x;
		this->filter_y = filter_y;
		this->filter_z = filter_z;
		this->filter_count = filter_x * filter_y * filter_z;
	}

	__device__ void compute_bounding_box(int &dim_x, int &dim_y, int &dim_z) {	

		vmin_x = vmin_y = vmin_z = 1e6f;
		vmax_x = vmax_y = vmax_z = -1e6f;
		for (int i = 0; i < points.size; ++i) {
			T x = points[3 * i + 0];
			T y = points[3 * i + 1];
			T z = points[3 * i + 2];

			vmin_x = min(vmin_x, x);
			vmin_y = min(vmin_y, y);
			vmin_z = min(vmin_z, z);

			vmax_x = max(vmax_x, x);
			vmax_y = max(vmax_y, y);
			vmax_z = max(vmax_z, z);
		}
		
		dim_x = (int)((vmax_x - vmin_x) / voxel_size) + 2; // padding to avoid numerical out of bound
		dim_y = (int)((vmax_y - vmin_y) / voxel_size) + 2;
		dim_z = (int)((vmax_z - vmin_z) / voxel_size) + 2;
	}
	
	void alloc(int dim_x, int dim_y, int dim_z) {
		this->dim_x = dim_x;
		this->dim_y = dim_y;
		this->dim_z = dim_z;

		// Mass allocation after bounding box is available
		int num_cells = dim_x * dim_y * dim_z;
		cell.resize(points.size);
		cell.zero();
		cell_count.resize(num_cells);
		cell_count.zero();
		cell_start.resize(num_cells + 1);
		cell_start.zero();
        tmp_cell_count.resize(num_cells);
		tmp_cell_count.zero();
		cell_sorted.resize(points.size);
		int num_points = points.size;
		neighbor_count.resize(num_points * filter_count);
		neighbor_count.zero();
	}

	__device__ void build() {
		
        int num_cells = dim_x * dim_y * dim_z;

		for (int i = 0; i < points.size; ++i) {
			T x = points[3 * i + 0];
			T y = points[3 * i + 1];
			T z = points[3 * i + 2];

			int cx = (int)((x - vmin_x) / voxel_size);
			int cy = (int)((y - vmin_y) / voxel_size);
			int cz = (int)((z - vmin_z) / voxel_size);

			cell[i] = (cz * dim_y + cy) * dim_x + cx;
		}

		for (int i = 0; i < points.size; ++i) {
			cell_count[cell[i]]++;
		}

		cell_start[0] = 0;
		for (int i = 1; i <= num_cells; ++i) {
			cell_start[i] = cell_start[i - 1] + cell_count[i - 1];
		}

        // Store point index in an order that allows query all points in a cell quickly
        for (int i = 0; i < points.size; ++i) {
            int k = cell[i];
            int h = tmp_cell_count[k];
            int offset = cell_start[k];
            cell_sorted[offset + h] = i;
            tmp_cell_count[k]++;
        }
	}

	__device__ void neighbor_brute_force(int i, T x, T y, T z, int filter_x, int filter_y, int filter_z, T voxel_size,
				  			 			 Query &query) 
	{
		// Center the filter at the current point
		T xmin = x - filter_x * 0.5 * voxel_size;
		T xmax = x + filter_x * 0.5 * voxel_size;
		T ymin = y - filter_y * 0.5 * voxel_size;
		T ymax = y + filter_y * 0.5 * voxel_size;
		T zmin = z - filter_z * 0.5 * voxel_size;
		T zmax = z + filter_z * 0.5 * voxel_size;

		for (int j = 0; j < points.size; ++j) {

			T vx = points[3 * j + 0];
			T vy = points[3 * j + 1];
			T vz = points[3 * j + 2];

			if (vx < xmin || vx > xmax || vy < ymin || vy > ymax || vz < zmin || vz > zmax) continue;

			// Determine which cell
			int fx = min(filter_x - 1, (int)((vx - xmin) / voxel_size));
			int fy = min(filter_y - 1, (int)((vy - ymin) / voxel_size));
			int fz = min(filter_z - 1, (int)((vz - zmin) / voxel_size));

			int f = (fz * filter_y + fy) * filter_x + fx;

			// Good point
			query(j, f, neighbor_count[i * filter_count + f]);
		}
	}

	/**
	 * Return the index of all points that fall within the filter centered at a query point
	 */
	__device__ void neighbor(int i, T x, T y, T z, int filter_x, int filter_y, int filter_z, T voxel_size,
				  			 Query &query) 
	{
		// Center the filter at the current point
		T xmin = x - filter_x * 0.5 * voxel_size;
		T xmax = x + filter_x * 0.5 * voxel_size;
		T ymin = y - filter_y * 0.5 * voxel_size;
		T ymax = y + filter_y * 0.5 * voxel_size;
		T zmin = z - filter_z * 0.5 * voxel_size;
		T zmax = z + filter_z * 0.5 * voxel_size;

		int nx = (int)((filter_x + 1) * 0.5);
		int ny = (int)((filter_y + 1) * 0.5);
		int nz = (int)((filter_z + 1) * 0.5);

		int center_x = (int)((x - vmin_x) / voxel_size);
		int center_y = (int)((y - vmin_y) / voxel_size);
		int center_z = (int)((z - vmin_z) / voxel_size);

		// We only need to check cells that intersect with this filter
		for (int oz = -nz; oz <= nz; ++oz) {
			for (int oy = -ny; oy <= ny; ++oy) {
				for (int ox = -nx; ox <= nx; ++ox) {
		
					int cx = center_x + ox;
					int cy = center_y + oy;
					int cz = center_z + oz;
                    
					if (cx < 0 || cx >= dim_x || cy < 0 || cy >= dim_y || cz < 0 || cz >= dim_z) continue;
					
					int cidx = (cz * dim_y + cy) * dim_x + cx;

					for (int k = cell_start[cidx]; k < cell_start[cidx + 1]; ++k) {

						int j = cell_sorted[k];

						T vx = points[3 * j + 0];
						T vy = points[3 * j + 1];
						T vz = points[3 * j + 2];

						if (vx < xmin || vx > xmax || vy < ymin || vy > ymax || vz < zmin || vz > zmax) continue;

						// Determine which cell
						int fx = min(filter_x - 1, (int)((vx - xmin) / voxel_size));
						int fy = min(filter_y - 1, (int)((vy - ymin) / voxel_size));
						int fz = min(filter_z - 1, (int)((vz - zmin) / voxel_size));

						int f = (fz * filter_y + fy) * filter_x + fx;

						// Good point
						query(j, f, neighbor_count[i * filter_count + f]);

					}
				}
			}
		}
	}

	/**
	 * Return the index of all points that fall within the filter centered at a query point
	 */
	__device__ void neighbor(int i, T x, T y, T z, int filter_x, int filter_y, int filter_z, T voxel_size,
							 int ox, int oy, int oz, /* the offset from the current cell */
				  			 Query &query) 
	{
		// Center the filter at the current point
		T xmin = x - filter_x * 0.5 * voxel_size;
		T xmax = x + filter_x * 0.5 * voxel_size;
		T ymin = y - filter_y * 0.5 * voxel_size;
		T ymax = y + filter_y * 0.5 * voxel_size;
		T zmin = z - filter_z * 0.5 * voxel_size;
		T zmax = z + filter_z * 0.5 * voxel_size;

		int nx = (int)((filter_x + 1) * 0.5);
		int ny = (int)((filter_y + 1) * 0.5);
		int nz = (int)((filter_z + 1) * 0.5);

		int center_x = (int)((x - vmin_x) / voxel_size);
		int center_y = (int)((y - vmin_y) / voxel_size);
		int center_z = (int)((z - vmin_z) / voxel_size);

		// We only need to check cells that intersect with this filter
		//for (int oz = -nz; oz <= nz; ++oz) {
		//	for (int oy = -ny; oy <= ny; ++oy) {
		//		for (int ox = -nx; ox <= nx; ++ox) {
		{{{
					int cx = center_x + ox;
					int cy = center_y + oy;
					int cz = center_z + oz;
                    
					//if (cx < 0 || cx >= dim_x || cy < 0 || cy >= dim_y || cz < 0 || cz >= dim_z) continue;
					if (cx < 0 || cx >= dim_x || cy < 0 || cy >= dim_y || cz < 0 || cz >= dim_z) return;

					int cidx = (cz * dim_y + cy) * dim_x + cx;

					for (int k = cell_start[cidx]; k < cell_start[cidx + 1]; ++k) {

						int j = cell_sorted[k];

						T vx = points[3 * j + 0];
						T vy = points[3 * j + 1];
						T vz = points[3 * j + 2];

						if (vx < xmin || vx > xmax || vy < ymin || vy > ymax || vz < zmin || vz > zmax) continue;

						// Determine which cell
						int fx = min(filter_x - 1, (int)((vx - xmin) / voxel_size));
						int fy = min(filter_y - 1, (int)((vy - ymin) / voxel_size));
						int fz = min(filter_z - 1, (int)((vz - zmin) / voxel_size));

						int f = (fz * filter_y + fy) * filter_x + fx;

						// Good point
						query(j, f, neighbor_count[i * filter_count + f]);

					}
				}
			}
		}
	}

	__device__ void build_neighbor_count(int i) {

			T x = points[3 * i + 0];
			T y = points[3 * i + 1];
			T z = points[3 * i + 2];

			// Center the filter at the current point
			T xmin = x - filter_x * 0.5 * voxel_size;
			T xmax = x + filter_x * 0.5 * voxel_size;
			T ymin = y - filter_y * 0.5 * voxel_size;
			T ymax = y + filter_y * 0.5 * voxel_size;
			T zmin = z - filter_z * 0.5 * voxel_size;
			T zmax = z + filter_z * 0.5 * voxel_size;
			int filter_count = filter_z * filter_y * filter_x;

			int nx = (int)((filter_x + 1) * 0.5);
			int ny = (int)((filter_y + 1) * 0.5);
			int nz = (int)((filter_z + 1) * 0.5);

			int center_x = (int)((x - vmin_x) / voxel_size);
			int center_y = (int)((y - vmin_y) / voxel_size);
			int center_z = (int)((z - vmin_z) / voxel_size);

			// We only need to check cells that intersect with this filter
			for (int oz = -nz; oz <= nz; ++oz) {
				for (int oy = -ny; oy <= ny; ++oy) {
					for (int ox = -nx; ox <= nx; ++ox) {
						int cx = center_x + ox;
						int cy = center_y + oy;
						int cz = center_z + oz;
						if (cx < 0 || cx >= dim_x || cy < 0 || cy >= dim_y || cz < 0 || cz >= dim_z) continue;
						int cidx = (cz * dim_y + cy) * dim_x + cx;

						for (int k = cell_start[cidx]; k < cell_start[cidx + 1]; ++k) {
							int j = cell_sorted[k];

							T vx = points[3 * j + 0];
							T vy = points[3 * j + 1];
							T vz = points[3 * j + 2];

							if (vx < xmin || vx > xmax || vy < ymin || vy > ymax || vz < zmin || vz > zmax) continue;

							// Determine which cell
							int fx = min(filter_x - 1, (int)((vx - xmin) / voxel_size));
							int fy = min(filter_y - 1, (int)((vy - ymin) / voxel_size));
							int fz = min(filter_z - 1, (int)((vz - zmin) / voxel_size));

							int f = (fz * filter_y + fy) * filter_x + fx;

							neighbor_count[i * filter_count + f]++;
						}
					}
				}
			}
		
	}

	void free() {
		cell.free();
		cell_sorted.free();
		cell_count.free();
		cell_start.free();
		tmp_cell_count.free();
		neighbor_count.free();
	}

};

template <typename T>
__global__ void kernelComputeGridBox(int batch_size, Array<GpuAlloc, Grid<GpuAlloc, T> > grids, Array<GpuAlloc, int> dims) {
	int b = blockIdx.x * blockDim.x + threadIdx.x;
	if (b >= batch_size) return;
	grids[b].compute_bounding_box(dims[b * 3 + 0], dims[b * 3 + 1], dims[b * 3 + 2]);
}

template <typename T>
__global__ void kernelBuildGrids(int batch_size, Array<GpuAlloc, Grid<GpuAlloc, T> > grids) {
	int b = blockIdx.x * blockDim.x + threadIdx.x;
	if (b >= batch_size) return;
	grids[b].build();
}

template <typename T>
__global__ void kernelBuildNeighborCount(int batch_size, int num_points, Array<GpuAlloc, Grid<GpuAlloc, T> > grids) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int b = idx / num_points;
	int i = idx % num_points;
	if (b >= batch_size) return;

	grids[b].build_neighbor_count(i);
}

template <typename T>
struct ForwardQuery : public Query {
    __device__ ForwardQuery(int i, const T *points,  const T *input, const T *filter, T *output, 
							int batch_size, int num_points, int filter_x, int filter_y, int filter_z, 
							int filter_c_in, int filter_c_out, T voxel_size) 
    : i(i), points(points), 
    input(input),
    filter(filter),
    output(output),
    batch_size(batch_size), num_points(num_points), filter_x(filter_x), filter_y(filter_y), filter_z(filter_z), 
    filter_c_in(filter_c_in), filter_c_out(filter_c_out), voxel_size(voxel_size)	
    {
		
    }

    __device__ void operator()(int ii, int f, int fsize) {            
		
		T inv_fsize = 1.0 / fsize;
		#pragma unroll
		for (int c = 0; c < filter_c_out; ++c) {
			#pragma unroll
			for (int k = 0; k < filter_c_in; ++k) {

				// Get filter weight
				T w = filter[(f * filter_c_in + k) * filter_c_out + c];

				output[i * filter_c_out + c] += w * input[ii * filter_c_in + k] * inv_fsize;
				//atomicAdd(&output[i * filter_c_out + c], w * input[ii * filter_c_in + k] * inv_fsize);
			}
		}      
    }

	int i;
    const T *points; const T *input; const T *filter; T *output; 
    int batch_size; int num_points; int filter_x; int filter_y; int filter_z; 
    int filter_c_in; int filter_c_out; float voxel_size;	
};

template <typename T>
__global__ void kernelForward(const T *points_flat, const T *input_flat, const T *filter, T *output_flat, 
							int batch_size, int num_points, int filter_x, int filter_y, int filter_z, 
							int filter_c_in, int filter_c_out, T voxel_size, 
							Array<GpuAlloc, Grid<GpuAlloc, T> > grids ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	/*
	int nx = (int)((filter_x + 1) / 2);
	int ny = (int)((filter_y + 1) / 2);
	int nz = (int)((filter_z + 1) / 2);
	int cells_x = nx * 2 + 1;
	int cells_y = ny * 2 + 1;
	int cells_z = nz * 2 + 1;
	int cells = cells_x * cells_y * cells_z;
	if (idx >= batch_size * num_points * cells) return;

	int offset = idx % cells;
	int ox = offset % cells_x; offset /= cells_x;
	int oy = offset % cells_y; offset /= cells_y;
	int oz = offset;
	ox -= nx;
	oy -= ny;
	oz -= nz;
	idx /= cells;
	*/

	int b = idx / num_points;
	int i = idx % num_points;
	if (b >= batch_size) return;
	
	const T *points = points_flat + b * num_points * 3;       // XYZ as input
	const T *input = input_flat + b * num_points * filter_c_in;
	T *output = output_flat + b * num_points * filter_c_out;

	Grid<GpuAlloc, T> &grid = grids[b]; 

	T x = points[3 * i + 0];
	T y = points[3 * i + 1];
	T z = points[3 * i + 2];

	ForwardQuery<T> query(i, points, input, filter, output, batch_size, num_points, filter_x, filter_y, filter_z, 
					      filter_c_in, filter_c_out, voxel_size);
	//grid.neighbor(i, x, y, z, filter_x, filter_y, filter_z, voxel_size, ox, oy, oz, query);
	
	// use grid
	//grid.neighbor(i, x, y, z, filter_x, filter_y, filter_z, voxel_size, query);

	grid.neighbor_brute_force(i, x, y, z, filter_x, filter_y, filter_z, voxel_size, query);
}

template <typename T> 
struct GradientQuery : Query {
	int j; T x, y, z;
	const T *grad_from_next_tensor;
	const T *points; const T *input; const T *filter; T *grad_input; T *grad_filter_thread_arr; 
    int batch_size; int num_points; int filter_x; int filter_y; int filter_z; int filter_count;
    int filter_c_in; int filter_c_out; float voxel_size;
	Array<GpuAlloc, int> &neighbor_count;

	__device__ GradientQuery(int j, T x, T y, T z, /* the point where we start the neighbor query */
							const T *grad_from_next_tensor, 
							const T *points, const T *input, const T *filter, T *grad_input, T *grad_filter_thread_arr,
							int batch_size, int num_points, int filter_x, int filter_y, int filter_z, 
							int filter_c_in, int filter_c_out, float voxel_size, Array<GpuAlloc, int> &neighbor_count) 
	: j(j), x(x), y(y), z(z),
	grad_from_next_tensor(grad_from_next_tensor),
    points(points), 
    input(input),
    filter(filter),
    grad_input(grad_input), grad_filter_thread_arr(grad_filter_thread_arr),
    batch_size(batch_size), num_points(num_points), filter_x(filter_x), filter_y(filter_y), filter_z(filter_z), 
    filter_c_in(filter_c_in), filter_c_out(filter_c_out), voxel_size(voxel_size), 
	neighbor_count(neighbor_count)
	{
		filter_count = filter_x * filter_y * filter_z;
	}

	__device__ void operator()(int ii, int f_ii, int fsize_ii) {
		
		// Take i as center
		T kx = points[3 * ii + 0];
		T ky = points[3 * ii + 1];
		T kz = points[3 * ii + 2];

		T xmin = kx - filter_x * 0.5 * voxel_size;
		T ymin = ky - filter_y * 0.5 * voxel_size;
		T zmin = kz - filter_z * 0.5 * voxel_size;

		// Check which cell the point pj is in w.r.t the point pi
		int fx = min(filter_x - 1, (int)((x - xmin) / voxel_size));
		int fy = min(filter_y - 1, (int)((y - ymin) / voxel_size));
		int fz = min(filter_z - 1, (int)((z - zmin) / voxel_size));

		int filter_index = (fz * filter_y + fy) * filter_x + fx;
		int count = neighbor_count[ii * filter_count + filter_index];
		if (count == 0) return; // FIXME: non-symmetric neighbor issue

		// For all types of filters
		#pragma unroll
		for (int c = 0; c < filter_c_out; ++c) {
			int out_index = ii * filter_c_out + c;
			T dL_dxi = grad_from_next_tensor[out_index];
			T dL_dxi_div_count = dL_dxi / (T)count;

			#pragma unroll
			for (int k = 0; k < filter_c_in; ++k) {
				int weight_index = (filter_index * filter_c_in + k) * filter_c_out + c;
				
				int in_index = j * filter_c_in + k;

				

				// Update the gradient of an input xi
				T w_as = filter[weight_index];
				grad_input[in_index] += dL_dxi_div_count * w_as;

				// Update the gradient of a filter weight
				T dxi_dw = input[in_index];
				//grad_filter_thread_arr[tid * num_weights + weight_index] += dL_dxi_div_count * dxi_dw;


				//grad_filter_thread_arr[weight_index] += dL_dxi_div_count * dxi_dw;

				atomicAdd(&grad_filter_thread_arr[weight_index], dL_dxi_div_count * dxi_dw);
			}
		}		
	}
};

template <typename T>
__global__ void kernelGradient(const T *grad_from_next_tensor_flat, 
								const T *points_flat,  const T *input_flat, const T *filter, 
								T *grad_input_flat, T *grad_filter_thread_arr, 
								int batch_size, int num_points, int filter_x, int filter_y, int filter_z, 
								int filter_c_in, int filter_c_out, T voxel_size, 
								Array<GpuAlloc, Grid<GpuAlloc, T> > grids) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	/*
	int nx = (int)((filter_x + 1) / 2);
	int ny = (int)((filter_y + 1) / 2);
	int nz = (int)((filter_z + 1) / 2);
	int cells_x = nx * 2 + 1;
	int cells_y = ny * 2 + 1;
	int cells_z = nz * 2 + 1;
	int cells = cells_x * cells_y * cells_z;
	if (idx >= batch_size * num_points * cells) return;

	int offset = idx % cells;
	int ox = offset % cells_x; offset /= cells_x;
	int oy = offset % cells_y; offset /= cells_y;
	int oz = offset;
	ox -= nx;
	oy -= ny;
	oz -= nz;
	idx /= cells;
	*/

	int b = idx / num_points;
	int j = idx % num_points;	
	if (b >= batch_size) return;
	
	const T *grad_from_next_tensor = grad_from_next_tensor_flat + b * num_points * filter_c_out;
	const T *points = points_flat + b * num_points * 3;       // XYZ as input
	const T *input = input_flat + b * num_points * filter_c_in;
	T *grad_input = grad_input_flat + b * num_points * filter_c_in;

	Grid<GpuAlloc, T> &grid = grids[b]; 
	
	T x = points[3 * j + 0];
	T y = points[3 * j + 1];
	T z = points[3 * j + 2];

	GradientQuery<T> query(j, x, y, z, grad_from_next_tensor, 
						   points, input, filter, grad_input, grad_filter_thread_arr, batch_size, num_points, filter_x, filter_y, filter_z, 
					       filter_c_in, filter_c_out, voxel_size, grid.neighbor_count);
	//grid.neighbor(j, x, y, z, filter_x, filter_y, filter_z, voxel_size, query);
	grid.neighbor_brute_force(j, x, y, z, filter_x, filter_y, filter_z, voxel_size, query);
}

/*
template <typename T>
__global__ void kernelReduction(T *grad_filter, Array<GpuAlloc, T> grad_filter_thread, int num_threads, int num_weights) {
	
	int weight_index = blockIdx.x * blockDim.x + threadIdx.x; 
	if (weight_index >= num_weights) return;
	
	for (int tid = 0; tid < num_threads; ++tid) {		
		grad_filter[weight_index] += grad_filter_thread[tid * num_weights + weight_index];		
	}

}*/

using namespace tensorflow;

REGISTER_OP("Conv3p")
    .Attr("T: {float , double}")
    .Input("points: T")
    .Input("input: T")
    .Input("filter: T")
    .Input("voxel_size: T")
    .Output("output: T")
    .Doc(R"doc(
Computes a 3-D convolution given a point cloud `input` and `filter` tensors.
The filter is applied at each point.
The filter size unit is voxel.
points: Shape `[batch, in_length, 3]`. Position of the point cloud.
input: Shape `[batch, in_length, in_channels]`. Other channels.
filter: Shape `[filter_depth, filter_height, filter_width, in_channels,
  out_channels]`. `in_channels` must match between `input` and `filter`.
voxel_size: size of a voxel to determine actual filter size w.r.t. the point cloud.
)doc");

REGISTER_OP("Conv3pGrad")
    .Attr("T: {float, double}")
    .Input("grad_from_next: T")
    .Input("points: T")
    .Input("input: T")
    .Input("filter: T")
    .Input("voxel_size: T")
    .Output("grad_input: T")
    .Output("grad_filter: T")
    .Doc(R"doc(
Computes the gradient of a 3-D point cloud convolution with respect to the input and weights
)doc");

template <typename T>
class Conv3pOp : public OpKernel {
 public:
  explicit Conv3pOp(OpKernelConstruction* context) : OpKernel(context) {
  }

  void Compute(OpKernelContext* context) override {
  	//long build_grid_time = 0;
    //long whole_time = 0;
  	//long convolution_time = 0;

  	//timer begin_whole_time;
    // Point tensor is of the following dimensions:
    // [ batch, num_points, 3 ]
    const Tensor& points_tensor = context->input(0);
    OP_REQUIRES(context, points_tensor.dims()==3, errors::InvalidArgument("Conv3p expects (batch_size, num_points, 3) points shape"));
    int batch_size = points_tensor.shape().dim_size(0);
    int num_points = points_tensor.shape().dim_size(1);
    auto points_flat = points_tensor.flat<T>();

    // Input tensor is of the following dimensions:
    const Tensor& input_tensor = context->input(1);
    OP_REQUIRES(context, input_tensor.shape().dim_size(0) == points_tensor.shape().dim_size(0), errors::InvalidArgument("Conv3p expects points and input tensor to have the same batch size"));
    OP_REQUIRES(context, input_tensor.shape().dim_size(1) == points_tensor.shape().dim_size(1), errors::InvalidArgument("Conv3p expects points and input tensor to have the same number of points"));
    int num_channels_in = input_tensor.shape().dim_size(2);
    auto input_flat = input_tensor.flat<T>();

    // Input filter is of the following dimensions:
    // [ filter_z, filter_y, filter_x, in_channels, out_channels]
    const Tensor& filter_tensor = context->input(2);
    int filter_z = filter_tensor.shape().dim_size(0);
    int filter_y = filter_tensor.shape().dim_size(1);
    int filter_x = filter_tensor.shape().dim_size(2);
    int filter_c_in = filter_tensor.shape().dim_size(3);
    int filter_c_out = filter_tensor.shape().dim_size(4);
    OP_REQUIRES(context, filter_c_in == num_channels_in, errors::InvalidArgument("Conv3p expects filter channels to be matched with input channels"));
    
    auto filter_flat = filter_tensor.flat<T>();    

    const Tensor& voxel_tensor = context->input(3);
    OP_REQUIRES(context, voxel_tensor.shape().dim_size(0) == 1, errors::InvalidArgument("Conv3p expects voxel tensor to have dimension 1."));
    const T *voxel_flat = &(voxel_tensor.flat<T>()(0));
	T voxel_size;	
	hipMemcpy(&voxel_size, voxel_flat, sizeof(T), hipMemcpyDeviceToHost);
	
    // Create output tensor
    Tensor* output_tensor = NULL;
    OP_REQUIRES_OK(context, context->allocate_output(0, TensorShape{batch_size, num_points, filter_c_out},
                                                     &output_tensor));
    auto output_flat = output_tensor->flat<T>();
    hipMemset(&(output_flat(0)), 0, sizeof(T) * batch_size * num_points * filter_c_out);

	// Launch a small kernel to build grid structure for all clouds in the batch
	//std::cout << "Building grid" << std::endl;
	//timer t1;
	std::vector<Grid<GpuAlloc, T> > cpu_grids(batch_size);
	for (int b = 0; b < batch_size; ++b) {
		const T *points = &(points_flat(0)) + b * num_points * 3;
		cpu_grids[b] = Grid<GpuAlloc, T>(Array<GpuAlloc, T>((T*)points, num_points), voxel_size, filter_x, filter_y, filter_z);
	}	
	Array<GpuAlloc, Grid<GpuAlloc, T> > grids;
	grids.resize(batch_size);
	hipMemcpy(grids.data, cpu_grids.data(), sizeof(Grid<GpuAlloc, T>) * batch_size, hipMemcpyHostToDevice);
	//std::cout << "t1: " << t1.getTimePassed() << std::endl;

	//timer t2;
	// Compute bounding box of each point cloud in parallel
	Array<GpuAlloc, int> dims; 
	dims.resize(batch_size * 3);

	dim3 blocks(divUp(batch_size, 32));
	dim3 threads(32);
	kernelComputeGridBox<<<blocks, threads>>>(batch_size, grids, dims);

	hipMemcpy(cpu_grids.data(), grids.data, sizeof(Grid<GpuAlloc, T>) * batch_size, hipMemcpyDeviceToHost);
	//std::cout << "t2: " << t2.getTimePassed() << std::endl;

	//timer t3;
	std::vector<int> cpu_dims(batch_size * 3);
	hipMemcpy(cpu_dims.data(), dims.data, sizeof(int) * batch_size * 3, hipMemcpyDeviceToHost);
	
	// After bounding box is computed, we can now allocate memory for grid structure
	for (int b = 0; b < batch_size; ++b) {
		int dim_x = cpu_dims[b * 3 + 0];
		int dim_y = cpu_dims[b * 3 + 1];
		int dim_z = cpu_dims[b * 3 + 2];
		cpu_grids[b].alloc(dim_x, dim_y, dim_z);
	}
	hipMemcpy(grids.data, cpu_grids.data(), sizeof(Grid<GpuAlloc, T>) * batch_size, hipMemcpyHostToDevice);
	//std::cout << "t3: " << t3.getTimePassed() << std::endl;

	//timer t4;
	// And then build cell information
	{
		dim3 blocks(divUp(batch_size, 32));
		dim3 threads(32);
		kernelBuildGrids<<<blocks, threads>>>(batch_size, grids);
	}
	{
		dim3 blocks(divUp(batch_size * num_points, kThreadsPerBlock));
		dim3 threads(kThreadsPerBlock);
		kernelBuildNeighborCount<<<blocks, threads>>>(batch_size, num_points, grids);
	}
	//hipDeviceSynchronize();
	//std::cout << "t4: " << t4.getTimePassed() << std::endl;


	timer t5;
	//std::cout << "Convolution" << std::endl;
	{
	int nx = (int)((filter_x + 1) / 2);
	int ny = (int)((filter_y + 1) / 2);
	int nz = (int)((filter_z + 1) / 2);
	int cells_x = nx * 2 + 1;
	int cells_y = ny * 2 + 1;
	int cells_z = nz * 2 + 1;
	int cells = cells_x * cells_y * cells_z;
	// Now parallelize over all batches and all points
	//dim3 blocks(divUp(batch_size * num_points * cells, kThreadsPerBlock));
	dim3 blocks(divUp(batch_size * num_points, kThreadsPerBlock));
	dim3 threads(kThreadsPerBlock);
	
	kernelForward<<<blocks, threads>>>(&(points_flat(0)), &(input_flat(0)), &(filter_flat(0)), &(output_flat(0)), 
										batch_size, num_points, filter_x, filter_y, filter_z, filter_c_in, filter_c_out, voxel_size, 
										grids);
	}
	hipDeviceSynchronize();
	std::cout << "conv time: " << t5.getTimePassed() << std::endl;

	//timer t6;
	for (int b = 0; b < batch_size; ++b) {
		cpu_grids[b].free();
	}
	grids.free();
	//std::cout << "Done" << std::endl;
	//std::cout << "t6: " << t6.getTimePassed() << std::endl;

	cudaErrorCheck(__LINE__);

    //whole_time = begin_whole_time.getTimePassed();
    //printf("time_for_convolution_fowardpass: %f\n", (double)convolution_time/1e9);
    //printf("time_for_build_grid_fowardpass: %f\n", (double)build_grid_time/1e9);
    //printf("whole_time_forwardpass: %f\n", (double)whole_time/1e9);
  }
};

#define REGISTER_GPU_KERNEL(T)                                  \
  REGISTER_KERNEL_BUILDER(                                      \
      Name("Conv3p").Device(DEVICE_GPU).TypeConstraint<T>("T"), \
      Conv3pOp<T>);
TF_CALL_float(REGISTER_GPU_KERNEL);
TF_CALL_double(REGISTER_GPU_KERNEL);
#undef REGISTER_GPU_KERNEL

////////////////////////////////////////////////////////////////////////////////
template <typename T>
class Conv3pGradOp : public OpKernel {
 public:
  explicit Conv3pGradOp(OpKernelConstruction* context) : OpKernel(context) {
  }

  void Compute(OpKernelContext* context) override {
    /**1. Setting things up **/
    // get the gradient tensor (from the later tensor)
    const Tensor& grad_from_next_tensor = context->input(0);
    auto grad_from_next_tensor_flat = grad_from_next_tensor.flat<T>();

    // get other inputs
    const Tensor& points_tensor = context->input(1);
    auto points_flat = points_tensor.flat<T>();

    const Tensor& input_tensor = context->input(2);
    auto input_flat = input_tensor.flat<T>();

    // infos about the inputs
    int batch_size = points_tensor.shape().dim_size(0);
    int num_points = points_tensor.shape().dim_size(1);

    // get the filters tensor (which include weights)
    const Tensor& filter_tensor = context->input(3);
    auto filter_flat = filter_tensor.flat<T>();
    
    const Tensor& voxel_tensor = context->input(4);
    OP_REQUIRES(context, voxel_tensor.shape().dim_size(0) == 1, errors::InvalidArgument("Conv3p expects voxel tensor to have dimension 1."));
    const T *voxel_flat = &(voxel_tensor.flat<T>()(0));
	T voxel_size;
	hipMemcpy(&voxel_size, voxel_flat, sizeof(T), hipMemcpyDeviceToHost);

    // dimensional infos for the filters tensor
    int filter_z = filter_tensor.shape().dim_size(0);
    int filter_y = filter_tensor.shape().dim_size(1);
    int filter_x = filter_tensor.shape().dim_size(2);
    int filter_c_in = filter_tensor.shape().dim_size(3);
    int filter_c_out = filter_tensor.shape().dim_size(4);
    int num_weights = filter_z * filter_y * filter_x * filter_c_in * filter_c_out;
    
    // Get shape of the grad tensors
    TensorShape grad_input_shape = input_tensor.shape();
    TensorShape grad_filter_shape = filter_tensor.shape();

    // Create the output tensor for the gradient of the inputs
    // How many points * number of input channel = how many gradients.
    Tensor* grad_input = NULL;
    OP_REQUIRES_OK(context, context->allocate_output(0, grad_input_shape, &grad_input));
    auto grad_input_flat = grad_input->flat<T>();
    hipMemset(&(grad_input_flat(0)), 0, sizeof(T) * input_tensor.shape().dim_size(0)*input_tensor.shape().dim_size(1)*input_tensor.shape().dim_size(2));

    // a) First we need to check if the size of the grad tensor and the number of points are compitable.
    OP_REQUIRES(context, grad_from_next_tensor.shape().dim_size(0) == batch_size, errors::InvalidArgument("backprop grad tensor has wrong size for dim 0"));
    OP_REQUIRES(context, grad_from_next_tensor.shape().dim_size(1) == num_points, errors::InvalidArgument("backprop grad tensor has wrong size for dim 1"));
    OP_REQUIRES(context, grad_from_next_tensor.shape().dim_size(2) == filter_c_out, errors::InvalidArgument("backprop grad tensor has wrong size for dim 2"));

    Tensor* grad_filter = NULL;
    OP_REQUIRES_OK(context, context->allocate_output(1, grad_filter_shape, &grad_filter));
    auto grad_filter_flat = grad_filter->flat<T>();
    hipMemset(&(grad_filter_flat(0)), 0, sizeof(T) * num_weights);
    
	// Launch a small kernel to build grid structure for all clouds in the batch
	//std::cout << "Building grid" << std::endl;

	cudaErrorCheck(__LINE__);

	timer t1;
	std::vector<Grid<GpuAlloc, T> > cpu_grids(batch_size);
	for (int b = 0; b < batch_size; ++b) {
		const T *points = &(points_flat(0)) + b * num_points * 3;
		cpu_grids[b] = Grid<GpuAlloc, T>(Array<GpuAlloc, T>((T*)points, num_points), voxel_size, filter_x, filter_y, filter_z);
	}	
	Array<GpuAlloc, Grid<GpuAlloc, T> > grids;
	grids.resize(batch_size);
	hipMemcpy(grids.data, cpu_grids.data(), sizeof(Grid<GpuAlloc, T>) * batch_size, hipMemcpyHostToDevice);

	// Compute bounding box of each point cloud in parallel
	Array<GpuAlloc, int> dims; 
	dims.resize(batch_size * 3);

	dim3 blocks(divUp(batch_size, 32));
	dim3 threads(32);
	kernelComputeGridBox<<<blocks, threads>>>(batch_size, grids, dims);

	hipMemcpy(cpu_grids.data(), grids.data, sizeof(Grid<GpuAlloc, T>) * batch_size, hipMemcpyDeviceToHost);

	std::vector<int> cpu_dims(batch_size * 3);
	hipMemcpy(cpu_dims.data(), dims.data, sizeof(int) * batch_size * 3, hipMemcpyDeviceToHost);
	
	// After bounding box is computed, we can now allocate memory for grid structure
	for (int b = 0; b < batch_size; ++b) {
		int dim_x = cpu_dims[b * 3 + 0];
		int dim_y = cpu_dims[b * 3 + 1];
		int dim_z = cpu_dims[b * 3 + 2];
		cpu_grids[b].alloc(dim_x, dim_y, dim_z);
	}
	hipMemcpy(grids.data, cpu_grids.data(), sizeof(Grid<GpuAlloc, T>) * batch_size, hipMemcpyHostToDevice);

	// And then build cell information
	{
		dim3 blocks(divUp(batch_size, 32));
		dim3 threads(32);
		kernelBuildGrids<<<blocks, threads>>>(batch_size, grids);
	}
	{
		dim3 blocks(divUp(batch_size * num_points, kThreadsPerBlock));
		dim3 threads(kThreadsPerBlock);
		kernelBuildNeighborCount<<<blocks, threads>>>(batch_size, num_points, grids);
	}

	hipDeviceSynchronize();
	std::cout << "gradient t1: " << t1.getTimePassed() << std::endl;

	cudaErrorCheck(__LINE__);

	timer t2;
	// Now parallelize over all batches and all points
	{
		/*
		int nx = (int)((filter_x + 1) / 2);
		int ny = (int)((filter_y + 1) / 2);
		int nz = (int)((filter_z + 1) / 2);
		int cells_x = nx * 2 + 1;
		int cells_y = ny * 2 + 1;
		int cells_z = nz * 2 + 1;
		int cells = cells_x * cells_y * cells_z;
		dim3 blocks(divUp(batch_size * num_points * cells, kThreadsPerBlock));
		*/
		dim3 blocks(divUp(batch_size * num_points, kThreadsPerBlock));
		dim3 threads(kThreadsPerBlock);
		kernelGradient<<<blocks, threads>>>(&(grad_from_next_tensor_flat(0)), &(points_flat(0)), &(input_flat(0)), &(filter_flat(0)),
											&(grad_input_flat(0)), &(grad_filter_flat(0)), // grad_filter_thread_arr.data,
											batch_size, num_points, filter_x, filter_y, filter_z, filter_c_in, filter_c_out, voxel_size, 
											grids);
	}
	hipDeviceSynchronize();
	std::cout << "gradient t2: " << t2.getTimePassed() << std::endl;
	
	cudaErrorCheck(__LINE__);
	/*
	// Reduction
	{
		dim3 blocks(divUp(num_weights, kThreadsPerBlock));
		dim3 threads(kThreadsPerBlock);
		kernelReduction<<<blocks, threads>>>(&(grad_filter_flat(0)), grad_filter_thread_arr, num_threads, num_weights);
	}
    grad_filter_thread_arr.free();	
	*/

	timer t3;
	for (int b = 0; b < batch_size; ++b) {
		cpu_grids[b].free();
	}
	grids.free();
	std::cout << "gradient t3: " << t3.getTimePassed() << std::endl;

    /**2. Compute gradient of the input**/
    // dL_dxj = (sum over xi that have xj as a neighbor)dL/dxi * w_as
    // w_as is the weight associated xi and xj. w_as = 0 if xj not contribute to xi.
    // we take avantaged of if xj is a neighbor of xi, then xi is also a neighbor of xi
    // or that we have symmetric neighborhood, then
    // dL_dxj = (sum over all xi that are neightbor of xj)dL/dxi * w_as

    // /**3. Compute gradient of the filter**/
    // Reminder: grad_from_next_tensor contain dL/dx, with xi is an output of the forward pass.
    // Reminder: we can calculate dL/dw = (sum over i)dL/dxi * dx/dw, with w is a weight that connect x to the next layer
    // =>
    // b) We do this by go through all the points and accumulate the gradients
    // Create the output tensor for the gradient of the filter

    // timer begin_backprop;
     //long time_backprop = begin_backprop.getTimePassed();
     //printf("time backprop : %f\n", (double)time_backprop / 1e9);
  }
};

#define REGISTER_GPU_KERNEL(T)                                        \
  REGISTER_KERNEL_BUILDER(                                            \
      Name("Conv3pGrad").Device(DEVICE_GPU).TypeConstraint<T>("T"),   \
      Conv3pGradOp<T>);
TF_CALL_float(REGISTER_GPU_KERNEL);
TF_CALL_double(REGISTER_GPU_KERNEL);
#undef REGISTER_GPU_KERNEL
